#include "hip/hip_runtime.h"
#include <stdio.h>		//host c code // then pass arguments to device code 
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <stdlib.h>
#include <sys/time.h>
#define min(x,y) ( ((x) < (y))? (x) : (y))


__global__
void kernel_syr2k(int N, int M, double *C, double *A, double *B){
  //int i, j, k;

  // for (k = 0; k < M; k++) {
    // for (i = 0; i < N; i++) {
	  // for (j = 0; j < min (N,i); j++) {
		// C[i][j] += A[j][k] * B[i][k] + B[j][k] * A[i][k];
	  // }
    // }
  // }
}


void init_array(int N, int M,  double *C,  double *A, double *B){
  int i, j;

  for (i = 0; i < N; i++)
    for (j = 0; j < M; j++) {
      A[i][j] = (double) (i*j%N) / N;
      B[i][j] = (double) (i*j%M) / M;
    }
  for (i = 0; i < N; i++)
    for (j = 0; j < N; j++)
      C[i][j] = (double) (i*j%N) / M;
}



void print_array(int N,
   double C[N][N])
{
  int i, j;

  fprintf(stderr, "==BEGIN DUMP_ARRAYS==\n");
  fprintf(stderr, "begin dump: %s", "C");
  for (i = 0; i < N; i++)
    for (j = 0; j < N; j++) {
 if ((i * N + j) % 20 == 0) fprintf (stderr, "\n");
 fprintf (stderr, "%0.2lf ", C[i][j]);
    }
  fprintf(stderr, "\nend   dump: %s\n", "C");
  fprintf(stderr, "==END   DUMP_ARRAYS==\n");
}


int main(int argc, char** argv)
{
  int N;
  int M;

  
  struct timeval t_start;
  struct timeval t_end;
  double etime;

  double* C;
  double* A;
  double* B;
  

  if (argc < 3) {
    printf("usage ./syr2k N M\n");
    return 0;
  }

  N = atoi(argv[1]);
  M = atoi(argv[2]);

  //hipMallocManaged(&C, N*N * sizeof(double));		//cuda allocation of unified Memory  
  //hipMallocManaged(&A, N*M * sizeof(double));
  //hipMallocManaged(&B, N*M * sizeof(double));
   C1 = (double*)malloc(N*N * sizeof(double));
   A1 = (double*)malloc(N*M * sizeof(double));
   B1 = (double*)malloc(N*M * sizeof(double));

  init_array (N, M, *((double(*)[N][N])C1), *((double(*)[N][M])A1), *((double(*)[N][M])B1));
  gettimeofday (&t_start, NULL);

  hipMallocManaged(&C, N*N * sizeof(double));		//cuda allocation of unified Memory  
  hipMallocManaged(&A, N*M * sizeof(double));
  hipMallocManaged(&B, N*M * sizeof(double));  
  
	
	int k = 0;
	for (int i = 0; i < N; i++){
		for (int j = 0; j < M; j++){
			A[k++] = A1[i, j];
			B[k++] = B1[i, j];
        }
	}
    int f = 0;
	for (int i = 0; i < N; i++){
		for (int j = 0; j < N; j++){
			C[f++] = C1[i, j];

		}
	}
	
  
  kernel_syr2k<<<1,256>>>(N, M, *((double(*)[N])C), *((double(*)[N])A), *((double(*)[N])B));
  hipDeviceSynchronize();
  gettimeofday (&t_end, NULL);

  etime = t_end.tv_sec - t_start.tv_sec + 
        (t_end.tv_usec - t_start.tv_usec) * 1.0e-6;

  print_array(N, *((double(*)[N][N])C));

  printf("execution time=%lf\n", etime);

  hipFree(C);		//freeing cuda data
  hipFree(A);
  hipFree(B);

  return 0;
}
